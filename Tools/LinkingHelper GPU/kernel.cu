
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <cstdlib>
#include <iostream>
#include <cstdint>
#include <fstream>
#include <stdlib.h>
#include <string>

using namespace std;

struct Test {
	char* Value;
	int Length;
};
struct Label {
	Test* Value;
	int Length;
};

Label* GetItems(int length);
Label* GetItems2(int length);


int GetTime() {
	return clock();
}

__global__ 
void vectorAdd( Label *items, Label *items2, int* result) {
	int i = blockIdx.x * blockDim.x;
	int j = threadIdx.x;
	
	int exists;
	int matches;
	int highestMatch = 0;
	matches = 0;
	exists = 0;
	
	if(items[i].Value[0].Length != items2[j].Value[0].Length)
		return;
	
	result[0] = 5;
	
	exists = 1;
	
	if(items[i].Value[0].Value[0] != items2[j].Value[0].Value[0]) {
		exists = 0;
	}
	//if(items[i].Value[0].Value[1] != items2[j].Value[0].Value[1]) {
	//	exists = 0;
	//}
	
	if(exists == 1){
		matches++;
	}
	
	//matches = matches * 100 / items[i].Length;
	
	highestMatch = matches;
	
	if(result[0] < highestMatch)
		result[0] = highestMatch;
}

#define M 512
int main( int argc, char *argv[]) {
	
	//int length = 1035;
	int length = 1000;
	int length2 = 1000;
	
	Label* items = GetItems(length);
	//Label* items2 = GetItems2(length2);
	Label* items2 = items;//GetItems(length2);
	cout << "Read complete." << endl;
	Label* _items;
	Label* _items2;
	int* _result;
	int* result = new int[1];
	int r = 0;
	
	result[0] = 0;
	
	cout << "Allocating memory on card." << endl;
	
	hipMalloc( (void**)&_items, sizeof(Label) * length );
	hipMalloc( (void**)&_items2, sizeof(Label) * length2 );
	hipMalloc( (void**)&_result, sizeof(int) );
	
	hipMemcpy( _items, items, length * sizeof(Label), hipMemcpyHostToDevice );
	hipMemcpy( _items2, items2, length2 * sizeof(Label), hipMemcpyHostToDevice );
	hipMemcpy( _result, result, sizeof(int), hipMemcpyHostToDevice );
	
	printf("Starting measuring\n");
	int startTime = GetTime();
	
	int exists;
	int matches;
	int highestMatch = 0;
	int w1 = 0;
	int w2 = 0;
	for(int i = 0; i < length; i++){
		highestMatch = 0;
		if(items[i].Length <= 0)
			continue;
		for(int j = 0; j < length; j++){
			if(items2[j].Length <= 0)
				continue;
			matches = 0;
			//for(int w1 = 0; w1 < items[i].Length; w1++){
				exists = 0;
				
				
				//for(int w2 = 0; w2 < items2[j].Length; w2++){
					if(items[i].Value[w1].Length != items2[j].Value[w2].Length)
						continue;
					
					if(items[i].Value[w1].Length == 0)
						continue;
					
					//cout << items[i].Value[w1].Value << ":" << items[i].Value[w1].Length << endl;
					//cout << items2[j].Value[w1].Value << ":" << items2[j].Value[w1].Length << endl;
					
					exists = 1;
					
					/*for(int c = 0; c < sizeof(items[i].Value[w1]); c++) {
						if(items[i].Value[w1][c] != items2[j].Value[w2][c]) {
							exists = 0;
						}
					}*/
					
					//cout << items[i].Value[w1].Value[0] << " " << items2[j].Value[w2].Value[0];
					
					if(items[i].Value[w1].Value[0] != items2[j].Value[w2].Value[0]) {
						exists = 0;
					}
					
				
				
				
					if(exists == 1){
						matches++;
						//cout << " " << exists << " " << matches << endl;
						//break;
					}
				//}
			//}
			
			//matches = matches * 100 / items[i].Length;
			//matches = matches * 100 / 2;
			
			if(matches > highestMatch)
				highestMatch = matches;
		}
	}
	r = highestMatch;
	
	int endTime = GetTime();
	
	printf("CPU: ");
	printf("%d\n", (endTime - startTime));
	
	startTime = GetTime();
	
	vectorAdd<<<length,length2>>>( _items, _items2, _result );
	
	endTime = GetTime();
	
	printf("GPU: ");
	printf("%d\n", (endTime - startTime));
	
	hipMemcpy( result, _result, sizeof(int), hipMemcpyDeviceToHost ) ;
	
	cout << r << endl;
	cout << result[0] << endl;
	
	// free the memory allocated on the GPU
	hipFree( _items );
	hipFree( _items2 );
	hipFree( _result );
	
	return 0;
}

Label* GetItems(int length) {
	string line;
	
	Label* items = new Label[length];
	
	ifstream afile;
	afile.open("Test1.txt", ios::in );
	
	int index = 0;
	string* words;	
	while ( getline (afile,line) )
    {
		Label lbl;
		words = new string[1000];
		//char *cstr = new char[line.length() + 1];
		//strcpy(cstr, line.c_str());
		int wordLength = 0;
		int startIndex = 0;
		int wordCount = 0;
		for(int i = 0; i < line.length(); i++) {
			if(i == (line.length() - 1))
				wordLength++;
			
			if(line[i] == ' ' || line[i] == '_' || i == (line.length() - 1)){
				
				if(wordLength == 0 || (startIndex + wordLength) >= line.length())
					continue;
				
				words[wordCount] = line.substr(startIndex, wordLength);
				
				wordLength = 0;
				startIndex= i + 1;
				wordCount++;
				continue;
			}
			wordLength++;
		}
		
		lbl.Value = new Test[wordCount];
		
		for(int i = 0; i < wordCount; i++) {
		
			Test test;
			
			char *cstr = new char[wordLength + 1];
			strcpy(cstr, words[i].c_str());
			
			test.Value = cstr;
			test.Length = words[i].length();
		
			lbl.Value[i] = test;
		}
		
		lbl.Length = wordCount;
		items[index++] = lbl;
    }
	
	afile.close();
	
	return items;
}
Label* GetItems2(int length) {
	string line;
	
	Label* items = new Label[length];
	
	ifstream afile;
	afile.open("Test2.txt", ios::in );
	
	int index = 0;
	string* words;
	while ( getline (afile,line) )
    {
		Label lbl;
		words = new string[1000];
		//char *cstr = new char[line.length() + 1];
		//strcpy(cstr, line.c_str());
		int wordLength = 0;
		int startIndex = 0;
		int wordCount = 0;
		for(int i = 0; i < line.length(); i++) {
			if(i == (line.length() - 1))
				wordLength++;
			
			if(line[i] == ' ' || line[i] == '_' || i == (line.length() - 1)){
				
				if(wordLength == 0 || (startIndex + wordLength) >= line.length())
					continue;
				
				words[wordCount] = line.substr(startIndex, wordLength);
				
				wordLength = 0;
				startIndex= i + 1;
				wordCount++;
				continue;
			}
			wordLength++;
		} 
		
		lbl.Value = new Test[wordCount];
		
		for(int i = 0; i < wordCount; i++) {
		
			Test test;
			
			char *cstr = new char[wordLength + 1];
			strcpy(cstr, words[i].c_str());
			
			test.Value = cstr;
			test.Length = words[i].length();
		
			lbl.Value[i] = test;
		}
		
		lbl.Length = wordCount;
		items[index++] = lbl;
    }
	
	afile.close();
	
	return items;
}